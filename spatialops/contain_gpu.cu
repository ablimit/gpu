
#include <hip/hip_runtime.h>
#include<iostream>
#include<fstream>
#include<time.h>
using namespace std;

typedef struct{
	int *x;
	int *y;
	int nr_vertice;
	int mbr[4];
	int *boxes;
}polygon;

__device__ int PtInPolygon(int x,int y, int *poly_x, int *poly_y, int nCount){
        int nCross=0,i;
        int x1,x2,y1,y2;
        double ix;
        for(i=0;i<nCount-1;i++){
                x1=poly_x[i];
                y1=poly_y[i];
                x2=poly_x[i+1];
                y2=poly_y[i+1];
                if(y1==y2)continue;
                if(y<min(y1,y2))continue;
                if(y>=max(y1,y2))continue;
                ix=(double)(y-y1)*(double)(x2-x1)/(double)(y2-y1)+x1;
                if(ix>x)nCross++;
        }
        return(nCross%2==1);
}


__global__ void kernel(int nr_v1, int *poly1_x, int *poly1_y, int nr_v2, int *poly2_x, int *poly2_y, int left, int top, int *result){
	int tid = threadIdx.x+blockIdx.x*blockDim.x+blockIdx.y*gridDim.x*blockDim.x;
        int x = blockIdx.x + left,y = blockIdx.y + top;
        int *poly_x, *poly_y, nr_v;
        poly_x = (tid%2 == 0) ? poly1_x:poly2_x;
	poly_y = (tid%2 == 0) ? poly1_y:poly2_y;
        nr_v = (tid%2 == 0) ? nr_v1:nr_v2;
        if(PtInPolygon(x, y, poly_x, poly_y, nr_v) == 1)
                result[tid] = 1;
        else result[tid] = 0;

}

void parsePoly(char *line,polygon *poly){
		int i, offset = 0;
		sscanf(line, "%d, %d %d %d %d", &poly->nr_vertice, &poly->mbr[0], &poly->mbr[1], &poly->mbr[2], &poly->mbr[3]);
		//printf("%d, %d %d %d %d\n", poly->nr_vertice, poly->mbr[0], poly->mbr[1], poly->mbr[2], poly->mbr[3]);
		while(line[offset++] != ',');
		while(line[offset++] != ',');
		poly->x = (int *)malloc(poly->nr_vertice*sizeof(int));
		poly->y = (int *)malloc(poly->nr_vertice*sizeof(int));
		for(i=0;i<poly->nr_vertice;i++){
			sscanf(line+offset, "%d %d", &poly->x[i], &poly->y[i]);
			while(line[offset++] != ',');
		}
}

int filter(polygon *poly1, polygon *poly2){
		/* Check whether the mbr of poly1 contains in poly2 */
		 if(poly2->mbr[0]<=poly1->mbr[0] && poly2->mbr[1]>=poly1->mbr[1] && poly2->mbr[2]<=poly1->mbr[2] && poly2->mbr[3]>=poly1->mbr[3])
			return 0;
		else return 1;
}

int main()
{
	static const int read_bufsize=65536;
        char polygon1[read_bufsize], polygon2[read_bufsize];
        const char *filename = "polygon";
        fstream polyfile;
        polyfile.open(filename,fstream::in | fstream::binary);
        polyfile.getline(polygon1,read_bufsize);
	polyfile.getline(polygon2,read_bufsize);

	polygon *poly1,*poly2;
	poly1 = (polygon *)malloc(sizeof(polygon));
	poly2 = (polygon *)malloc(sizeof(polygon));
	parsePoly(polygon1, poly1);
	parsePoly(polygon2, poly2);
	if(filter(poly1,poly2)){
		cout<<"NO!"<<endl;
		return 1;
	}
	int *dev_poly1_x, *dev_poly1_y, *dev_poly2_x, *dev_poly2_y;
	int *host_result, *dev_result;
	int boxsize = (poly1->mbr[1]-poly1->mbr[0]+1)*(poly1->mbr[3]-poly1->mbr[2]+1);
	hipMalloc((void **)&dev_poly1_x, poly1->nr_vertice*sizeof(int));
	hipMalloc((void **)&dev_poly1_y, poly1->nr_vertice*sizeof(int));
	hipMalloc((void **)&dev_poly2_x, poly2->nr_vertice*sizeof(int));
	hipMalloc((void **)&dev_poly2_y, poly2->nr_vertice*sizeof(int));
	hipMalloc((void **)&dev_result, 2*boxsize*sizeof(int));

	hipMemcpy(dev_poly1_x, poly1->x, poly1->nr_vertice*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_poly1_y, poly1->y, poly1->nr_vertice*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_poly2_x, poly2->x, poly2->nr_vertice*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_poly2_y, poly2->y, poly2->nr_vertice*sizeof(int), hipMemcpyHostToDevice);

	clock_t start, end;
	start = clock();
	dim3 grids(poly1->mbr[1]-poly1->mbr[0],poly1->mbr[3]-poly1->mbr[2]);
	kernel<<<grids, 2>>>(poly1->nr_vertice, dev_poly1_x, dev_poly1_y, poly2->nr_vertice, dev_poly2_x, dev_poly2_y, poly1->mbr[0], poly1->mbr[2], dev_result);

	host_result = (int *)malloc(2*boxsize*sizeof(int));
	hipMemcpy(host_result, dev_result, 2*boxsize*sizeof(int), hipMemcpyDeviceToHost);
	for(int i=0;i<boxsize;i++){
	//	cout<<i % (poly1->mbr[1]-poly1->mbr[0])+poly1->mbr[0]<<" "<<i/(poly1->mbr[1]-poly1->mbr[0])+poly1->mbr[2]<<endl;
		if(host_result[2*i] == 1 && host_result[2*i+1] == 0){
			end = clock();
			cout<<"NO! Time used: "<<end-start<<endl;
			return 1;
		}
	}
	end = clock();
	cout<<"YES! Time used: "<<end-start<<endl;
	return 0;
}
