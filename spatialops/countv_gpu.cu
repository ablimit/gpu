#include "hip/hip_runtime.h"
#include<stdio.h>
#include<string.h>
#include<sqludf.h>

/* if poly[blockIdx.x]=',', set result[blockIdx.x]=1. The number of vertices is the number of 1 in result plus 1 */
__global__ void kernel(char *poly, int *result, int len){
	if(blockIdx.x<len){
		if(poly[blockIdx.x]==',')
			result[blockIdx.x] = 1;
		else result[blockIdx.x] = 0;
	}
}

void SQL_API_FN countv(
	SQLUDF_VARCHAR *polygon, 
	SQLUDF_INTEGER *result, 
	SQLUDF_NULLIND *nullpolygon,
	SQLUDF_NULLIND *nullresult, 
	SQLUDF_TRAIL_ARGS)
{
	char * dev_poly;
	int * dev_result;
	int * host_result;
	int i;
	int len = strlen(polygon);
	hipMalloc((void **)&dev_poly, len);
	hipMalloc((void **)&dev_result, len*sizeof(int));
	
	hipMemcpy(dev_poly, polygon, len, hipMemcpyHostToDevice);
	kernel<<<len,1>>> (dev_poly, dev_result, len);
	host_result = (int *)malloc(len*sizeof(int));
	hipMemcpy(host_result, dev_result, len*sizeof(int), hipMemcpyDeviceToHost);

	//count the 1's in host_result
	*result = 0;
	for(i=0;i<len;i++){
		if(host_result[i]==1)*result++;
	}
	*result ++;	
	*nullresult = 0;
	return;
}
