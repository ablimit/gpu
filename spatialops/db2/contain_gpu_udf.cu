#include "hip/hip_runtime.h"
#include<stdio.h>
#include<sqludf.h>

typedef struct{
	int *x;
	int *y;
	int nr_vertice;
	int mbr[4];
	int *boxes;
}polygon;

__device__ int PtInPolygon(int x,int y, int *poly_x, int *poly_y, int nCount){
        int nCross=0,i;
        int x1,x2,y1,y2;
        double ix;
        for(i=0;i<nCount-1;i++){
                x1=poly_x[i];
                y1=poly_y[i];
                x2=poly_x[i+1];
                y2=poly_y[i+1];
                if(y1==y2)continue;
                if(y<min(y1,y2))continue;
                if(y>=max(y1,y2))continue;
                ix=(double)(y-y1)*(double)(x2-x1)/(double)(y2-y1)+x1;
                if(ix>x)nCross++;
        }
        return(nCross%2==1);
}


__global__ void kernel(int nr_v1, int *poly1_x, int *poly1_y, int nr_v2, int *poly2_x, int *poly2_y, int left, int top, int *result){
	int tid = threadIdx.x+blockIdx.x*blockDim.x+blockIdx.y*gridDim.x*blockDim.x;
        int x = blockIdx.x + left,y = blockIdx.y + top;
        int *poly_x, *poly_y, nr_v;
        poly_x = (tid%2 == 0) ? poly1_x:poly2_x;
	poly_y = (tid%2 == 0) ? poly1_y:poly2_y;
        nr_v = (tid%2 == 1) ? nr_v1:nr_v2;
        if(PtInPolygon(x, y, poly_x, poly_y, nr_v) == 1)
                result[tid] = 1;
        else result[tid] = 0;

}

void parsePoly(char *line,polygon *poly){
		int i, offset = 0;
		sscanf(line, "%d, %d %d %d %d", &poly->nr_vertice, &poly->mbr[0], &poly->mbr[1], &poly->mbr[2], &poly->mbr[3]);
		poly->x = (int *)malloc(poly->nr_vertice*sizeof(int));
		poly->y = (int *)malloc(poly->nr_vertice*sizeof(int));
		while(line[offset++] != ',');
		while(line[offset++] != ',');
		for(i=0;i<poly->nr_vertice;i++){
			sscanf(line+offset, "%d %d", &poly->x[i], &poly->y[i]);
			while(line[offset++] != ',');
		}

}

int filter(polygon *poly1, polygon *poly2){
		/* Check whether the mbr of poly1 contains in poly2 */
		 if(poly2->mbr[0]<=poly1->mbr[0] && poly2->mbr[1]>=poly1->mbr[1] && poly2->mbr[2]<=poly1->mbr[2] && poly2->mbr[3]>=poly1->mbr[3])
			return 0;
		else return 1;
}

void SQL_API_FN contain(
	SQLUDF_VARCHAR *polygon1, 
	SQLUDF_VARCHAR *polygon2, 
	SQLUDF_INTEGER *result, 
	SQLUDF_NULLIND *nullpolygon1, 
	SQLUDF_NULLIND *nullpolygon2, 
	SQLUDF_NULLIND *nullresult, 
	SQLUDF_TRAIL_ARGS)
{
	polygon *poly1,*poly2;
	FILE *fp = fopen("/home/xxu37/gpuproject/contain/debug.txt", "w+");
	poly1 = (polygon *)malloc(sizeof(polygon));
	poly2 = (polygon *)malloc(sizeof(polygon));
	
	parsePoly(polygon1, poly1);
	parsePoly(polygon2, poly2);
	if(filter(poly1,poly2)){
		*result = 0;
		*nullresult = 0;
		return;
	}
	int *dev_poly1_x, *dev_poly1_y, *dev_poly2_x, *dev_poly2_y;
	int *host_result, *dev_result;	
	int boxsize = (poly1->mbr[1]-poly1->mbr[0]+1)*(poly1->mbr[3]-poly1->mbr[2]+1);
	fprintf(fp, "%s\n", "before cuda ok!");
	hipMalloc((void **)&dev_poly1_x, poly1->nr_vertice*sizeof(int));
	hipMalloc((void **)&dev_poly1_y, poly1->nr_vertice*sizeof(int));
	hipMalloc((void **)&dev_poly2_x, poly2->nr_vertice*sizeof(int));
	hipMalloc((void **)&dev_poly2_y, poly2->nr_vertice*sizeof(int));
	hipMalloc((void **)&dev_result, 2*boxsize*sizeof(int));
	fprintf(fp, "%s\n", "hipMalloc ok!");
	hipMemcpy(dev_poly1_x, poly1->x, poly1->nr_vertice*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_poly1_y, poly1->y, poly1->nr_vertice*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_poly2_x, poly2->x, poly2->nr_vertice*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_poly2_y, poly2->y, poly2->nr_vertice*sizeof(int), hipMemcpyHostToDevice);
	fprintf(fp, "%s\n", "hipMemcpy ok!");
	dim3 grids(poly1->mbr[1]-poly1->mbr[0],poly1->mbr[3]-poly1->mbr[2]);
	kernel<<<grids, 2>>>(poly1->nr_vertice, dev_poly1_x, dev_poly1_y, poly2->nr_vertice, dev_poly2_x, dev_poly2_y, poly1->mbr[0], poly1->mbr[2], dev_result);
	fprintf(fp, "%s\n", "kernel ok!");
	host_result = (int *)malloc(2*boxsize*sizeof(int));
	hipMemcpy(host_result, dev_result, 2*boxsize*sizeof(int), hipMemcpyDeviceToHost);
	for(int i=0;i<boxsize;i++){
	//	cout<<i % (poly1->mbr[1]-poly1->mbr[0])+poly1->mbr[0]<<" "<<i/(poly1->mbr[1]-poly1->mbr[0])+poly1->mbr[2]<<endl;
		if(host_result[2*i] == 1 && host_result[2*i+1] == 0){
			*result = 0;
			*nullresult = 0;
			return;
		}
	}
	*result = 1;
	*nullresult = 0;
	return;
}
