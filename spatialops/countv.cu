
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<string.h>

/* if poly[blockIdx.x]=',', set result[blockIdx.x]=1. The number of vertices is the number of 1 in result plus 1 */
__global__ void kernel(char *poly, int *result, int len){
        if(blockIdx.x<len){
                if(poly[blockIdx.x]==',')
                        result[blockIdx.x] = 1;
                else result[blockIdx.x] = 0;
        }
}

int countv(char *polygon)
{
        char * dev_poly;
        int * dev_result;
        int * host_result;
        int i,result;
        int len = strlen(polygon);
        hipMalloc((void **)&dev_poly, len);
        hipMalloc((void **)&dev_result, len*sizeof(int));

        hipMemcpy(dev_poly, polygon, len, hipMemcpyHostToDevice);
        kernel<<<len,1>>> (dev_poly, dev_result, len);
        host_result = (int *)malloc(len*sizeof(int));
        hipMemcpy(host_result, dev_result, len*sizeof(int), hipMemcpyDeviceToHost);

        //count the 1's in host_result
        result = 0;
        for(i=0;i<len;i++){
                if(host_result[i]==1)result++;
        }
        result ++;
        return result;
}

